#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int *a, *b;  // host data
int *c, *c2;  // result

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}

//GPU kernel 
__global__
void convGPU(int *A, int *B, int *C, int N){
    //Get current column and row
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int element = 0;
    //Verify if it's in the bounds
    if (i < (N+N-1)){
        //Compute an element
        for(int k = 0; k < N; k++){
            element += A[k] * (i - k < N && i - k >= 0 ? B[i - k] : 0);
        }
    }
    //Store result of the element
    C[i] = element;
}

//CPU function
void convCPU(int *A, int *B, int *C, int N){
    for (int i = 0; i < N+N - 1; ++i)
    {
        int element = 0;
        for (int k = 0; k < N; ++k)
        {
            element += A[k] * (i - k < N && i - k >= 0 ? B[i - k] : 0);
        }
        C[i] = element;
    }
}

int main(int argc,char **argv)
{
    printf("Begin \n");
    //Size of signals
    int n=150000;
    //iterations
    int m = n+n - 1;
    //Number of blocks
    int nBytes = n*sizeof(int);
    int cBytes = m*sizeof(int);
    //Block size and number
    int block_size, block_no;

    //memory allocation 
    a = (int *) malloc(nBytes);
    b = (int *) malloc(nBytes);
    c = (int *) malloc(cBytes);
    c2 = (int *) malloc(cBytes);

    int *a_d,*b_d,*c_d;
    block_size = 350; //threads per block 
    block_no = (m+1)/block_size;
    
    //Work definition
    dim3 dimBlock(block_size, 1, 1);
    dim3 dimGrid(block_no, 1, 1);

    // Data filling
    for(int i=0;i<n;i++)
        a[i]=i,b[i]=i;

    printf("\n\nAllocating device memory on host..\n");
   //GPU memory allocation
    hipMalloc((void **) &a_d, n*sizeof(int));
    hipMalloc((void **) &b_d, n*sizeof(int));
    hipMalloc((void **) &c_d, m*sizeof(int));

    printf("Copying to device..\n");
    hipMemcpy(a_d, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(int), hipMemcpyHostToDevice);

    //Starting clock
    clock_t start_d=clock();
    printf("Doing GPU convolution\n\n");
    convGPU<<<block_no,block_size>>>(a_d, b_d, c_d, n);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    clock_t end_d = clock();
    
    printf("Doing CPU convolution\n");
    clock_t start_h = clock();
    convCPU(a, b, c2, n);
    clock_t end_h = clock();
    
    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    hipMemcpy(c, c_d, m*sizeof(int), hipMemcpyDeviceToHost);
    printf("m = %d \t GPU time = %fs \t CPU time = %fs\n", m, time_d, time_h);

    //Free GPU memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}
